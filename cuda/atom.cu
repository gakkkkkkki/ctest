
#include <hip/hip_runtime.h>
#if __CUDA_ARCH__ < 600
  #define atomicMin_block(X,Y) atomicMin(X,Y)
  #define atomicAdd_block(X,Y) atomicAdd(X,Y)
#endif


__global__
void glo(int * x, int * y) {
   atomicAdd(x+3,1);
}

__global__
void blo(int * x, int *  y) {
   atomicAdd_block(x+3,1);
}

__global__
void sha(int * x, int *  y) {
   __shared__ int c[1024];
   atomicAdd(c+3,1);
   x[3] = c[3];
}


__global__
void shablo(int * x, int *  y) {
   __shared__ int c[1024];
   atomicAdd_block(c+3,1);
   x[3] = c[3];
}

